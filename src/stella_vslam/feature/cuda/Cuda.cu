#include <hip/hip_runtime_api.h>
#include "stella_vslam/feature/cuda/Cuda.hpp"

namespace stella_vslam {
namespace feature {
namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} // namespace cuda
} // namespace feature
} // namespace stella_vslam